#include "hip/hip_runtime.h"
#include "ethash_cu_miner_kernel.h"

void run_ethash_hash(
	__global__ hash32_t* g_hashes,
	__constant__ hash32_t const* g_header,
	__global__ hash128_t const* g_dag,
	ulong start_nonce,
	uint isolate
) 
{
}

void run_ethash_search(
	__global__ volatile uint* restrict g_output,
	__constant__ hash32_t const* g_header,
	__global__ hash128_t const* g_dag,
	ulong start_nonce,
	ulong target,
	uint isolate
)
{
